#include "hip/hip_runtime.h"
// Copyright CERN and copyright holders of ALICE O2. This software is
// distributed under the terms of the GNU General Public License v3 (GPL
// Version 3), copied verbatim in the file "COPYING".
//
// See http://alice-o2.web.cern.ch/license for full licensing information.
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

/// \file testGPUsortCUDA.cu
/// \author ...

#define GPUCA_GPUTYPE_PASCAL

#define BOOST_TEST_MODULE Test GPUCommonAlgorithm Sorting CUDA
#define BOOST_TEST_MAIN
#define BOOST_TEST_DYN_LINK

#include <iostream>
#include <cstring>
#include <boost/test/unit_test.hpp>
#include "GPUCommonAlgorithmCUDA.cuh"

///////////////////////////////////////////////////////////////
// Test setup and tear down
///////////////////////////////////////////////////////////////

struct TestEnvironment {
  TestEnvironment() : size(101), data(nullptr), sorted(size), tolerance(10 * std::numeric_limits<float>::epsilon())
  {
    const auto cudaErrorCode = hipMallocManaged(&data, size * sizeof(float));
    if (cudaErrorCode != hipSuccess) {
      std::cerr << "ErrorCode " << cudaErrorCode << " " << hipGetErrorName(cudaErrorCode) << ": " << hipGetErrorString(cudaErrorCode) << std::endl;
    }

    // create an array of unordered floats with negative values
    for (size_t i = 0; i < size; i++) {
      data[i] = size / 2 - i;
    }
    // create copy
    std::memcpy(sorted.data(), data, size * sizeof(float));
    // sort
    std::sort(sorted.begin(), sorted.end());
  }

  ~TestEnvironment()
  {
    hipFree(data);
  };

  const size_t size;
  float* data;
  std::vector<float> sorted;
  const float tolerance;
};

///////////////////////////////////////////////////////////////

__global__ void sortInThread(float* data, size_t dataLength)
{
  // make sure only one thread is working on this.
  if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    o2::gpu::CAAlgo::sort(data, data + dataLength);
  }
}

__global__ void sortInThreadWithOperator(float* data, size_t dataLength)
{
  // make sure only one thread is working on this.
  if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    o2::gpu::CAAlgo::sort(data, data + dataLength, [](float a, float b) { return a < b; });
  }
}

///////////////////////////////////////////////////////////////

__global__ void sortInBlock(float* data, size_t dataLength)
{
  o2::gpu::CAAlgo::sortInBlock(data, data + dataLength);
}

__global__ void sortInBlockWithOperator(float* data, size_t dataLength)
{
  o2::gpu::CAAlgo::sortInBlock(data, data + dataLength, [](float a, float b) { return a < b; });
}
///////////////////////////////////////////////////////////////

BOOST_AUTO_TEST_SUITE(TestsortInThread)

BOOST_FIXTURE_TEST_CASE(GPUsortThreadCUDA, TestEnvironment)
{
  sortInThread<<<1, 1>>>(data, size);
  const auto cudaErrorCode = hipDeviceSynchronize();
  BOOST_CHECK_EQUAL(cudaErrorCode, hipSuccess);
  if (cudaErrorCode != hipSuccess) {
    std::cerr << "ErrorCode " << cudaErrorCode << " " << hipGetErrorName(cudaErrorCode) << ": " << hipGetErrorString(cudaErrorCode) << std::endl;
  }

  for (size_t i = 0; i < size; i++) {
    if (std::fabs(sorted[i]) < tolerance) {
      BOOST_CHECK_SMALL(data[i], tolerance);
    } else {
      BOOST_CHECK_CLOSE(sorted[i], data[i], 1.0 / tolerance);
    }
  }
}

BOOST_FIXTURE_TEST_CASE(GPUsortThreadOperatorCUDA, TestEnvironment)
{
  sortInThreadWithOperator<<<1, 1>>>(data, size);
  const auto cudaErrorCode = hipDeviceSynchronize();
  BOOST_CHECK_EQUAL(cudaErrorCode, hipSuccess);
  if (cudaErrorCode != hipSuccess) {
    std::cerr << "ErrorCode " << cudaErrorCode << " " << hipGetErrorName(cudaErrorCode) << ": " << hipGetErrorString(cudaErrorCode) << std::endl;
  }

  for (size_t i = 0; i < size; i++) {
    if (std::fabs(sorted[i]) < tolerance) {
      BOOST_CHECK_SMALL(data[i], tolerance);
    } else {
      BOOST_CHECK_CLOSE(sorted[i], data[i], 1.0 / tolerance);
    }
  }
}

BOOST_AUTO_TEST_SUITE_END()

BOOST_AUTO_TEST_SUITE(TestsortInBlock)

BOOST_FIXTURE_TEST_CASE(GPUsortBlockCUDA, TestEnvironment)
{
  sortInBlock<<<1, 128>>>(data, size);
  const auto cudaErrorCode = hipDeviceSynchronize();
  BOOST_CHECK_EQUAL(cudaErrorCode, hipSuccess);
  if (cudaErrorCode != hipSuccess) {
    std::cerr << "ErrorCode " << cudaErrorCode << " " << hipGetErrorName(cudaErrorCode) << ": " << hipGetErrorString(cudaErrorCode) << std::endl;
  }

  for (size_t i = 0; i < size; i++) {
    if (std::fabs(sorted[i]) < tolerance) {
      BOOST_CHECK_SMALL(data[i], tolerance);
    } else {
      BOOST_CHECK_CLOSE(sorted[i], data[i], 1.0 / tolerance);
    }
  }
}

BOOST_FIXTURE_TEST_CASE(GPUsortBlockOperatorCUDA, TestEnvironment)
{
  sortInBlockWithOperator<<<1, 128>>>(data, size);
  const auto cudaErrorCode = hipDeviceSynchronize();
  BOOST_CHECK_EQUAL(cudaErrorCode, hipSuccess);
  if (cudaErrorCode != hipSuccess) {
    std::cerr << "ErrorCode " << cudaErrorCode << " " << hipGetErrorName(cudaErrorCode) << ": " << hipGetErrorString(cudaErrorCode) << std::endl;
  }

  for (size_t i = 0; i < size; i++) {
    if (std::fabs(sorted[i]) < tolerance) {
      BOOST_CHECK_SMALL(data[i], tolerance);
    } else {
      BOOST_CHECK_CLOSE(sorted[i], data[i], 1.0 / tolerance);
    }
  }
}

BOOST_AUTO_TEST_SUITE_END()
